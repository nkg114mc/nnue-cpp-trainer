#include "hip/hip_runtime.h"
#include "featrans_kernel.cuh"
#include <stdio.h>

//typedef unsigned int uint32_t;
//typedef int int32_t;

/*
__global__ void test_kernel(void) {}

namespace Wrapper {
	void wrapper(void)
	{
		test_kernel <<<1, 1>>> ();
		printf("Hello, world!");
	}
}
*/

/*
def _find_nearest_divisor(value, target):
    divisors = []
    for i in range(1, value + 1):
        if value % i == 0:
            divisors.append((i, abs(target - i)))
    divisors.sort(key=lambda x: x[1])
    return divisors[0][0]


_num_threads_forward_cache = dict()


def _get_num_threads_for_forward(output_size):
    optimal_num_threads = 512
    if output_size not in _num_threads_forward_cache:
        _num_threads_forward_cache[output_size] = _find_nearest_divisor(output_size, optimal_num_threads)

    return _num_threads_forward_cache[output_size]
*/

int32_t get_num_threads_for_forward(int32_t output_size) {
    return 256;
}

int32_t get_num_threads_for_backward(int32_t output_size) {
    return 256;
}

__global__
/*
    @assumptions:
        The blocks must have dimensionality (BATCH_SIZE,)
        The threads must have dimensionality (N,), where
        N * output_thread_slice_size == output_size.

    @param: feature_indices
        A matrix of shape (BATCH_SIZE, max_active_features)
        containing indices of active features for each position
        in a batch. Feature index of -1 means that the slot is empty
        and the weights will not be accumulated for it. Moreover
        no further indices from this block will be considered.
        The indices form an implicit matrix of shape
        (BATCH_SIZE, NUM_INPUTS), where the first dimension index is
        inferred from the memory location (BATCH_SIZE), and the
        second dimension index is stored in the feature_indices matrix.
        The type for feature indices is int32_t.

    @param: feature_values
        A matrix of shape (BATCH_SIZE, max_active_features)
        containing the values (arity) of the corresponding
        feature index in feature_indices.
        The type for the feature value (arity) is float32.

    @param: weight
        The weight matrix of shape (NUM_INPUTS, output_size).
        Weights must be of type float32.

    @param: bias
        The bias vector of shape (output_size,).
        Bias values must be of type float32.

    @param: output
        An output matrix of shape (BATCH_SIZE, output_size).
        It may not be initialized, bias is always copied
        to the output first.
        Output values must have type float32.
*/
void feature_transformer_slice_forward(
    const int32_t* const feature_indices,
    const float*   const feature_values,
    const float*   const weight,
    const float*   const bias,
          float*   const output
) {

    const uint32_t output_size = 256;
    const uint32_t max_active_features = 32; 
    const uint32_t output_thread_slice_size = 1;

    __shared__
          float          shared_output[output_size];

    const uint32_t       block_idx           = blockIdx.x;
    const uint32_t       slice_offset        = threadIdx.x * output_thread_slice_size;

          float*   const output_slice        = output + block_idx * output_size + slice_offset;
    const float*   const bias_slice          = bias                               + slice_offset;
          float*         shared_output_slice = shared_output                      + slice_offset;

    const int32_t* const feature_index_row   = feature_indices + block_idx * max_active_features;
    const float*   const feature_value_row   = feature_values  + block_idx * max_active_features;

    #pragma unroll
    for (uint32_t s = 0; s < output_thread_slice_size; ++s)
    {
        shared_output_slice[s] = bias_slice[s];
    }

    for (uint32_t k = 0; k < max_active_features; ++k)
    {
        const int32_t feature_index = feature_index_row[k];
        const float   feature_value = feature_value_row[k];
        if (feature_index != -1)
        {
            const float* const weight_slice = weight + feature_index * output_size + slice_offset;
            #pragma unroll
            for (uint32_t s = 0; s < output_thread_slice_size; ++s)
            {
                shared_output_slice[s] += weight_slice[s] * feature_value;
            }
        } else break;
    }

    #pragma unroll
    for (uint32_t s = 0; s < output_thread_slice_size; ++s)
    {
        output_slice[s] = shared_output_slice[s];
    }
}


__global__
/*
    @assumptions:
        The blocks must have dimensionality (BATCH_SIZE,)
        The threads must have dimensionality (N,), where
        N * output_thread_slice_size == output_size.

    @param: feature_indices
        A matrix of shape (BATCH_SIZE, max_active_features)
        containing indices of active features for each position
        in a batch. Feature index of -1 means that the slot is empty
        and the weights will not be accumulated for it. Moreover
        no further indices from this block will be considered.
        The indices form an implicit matrix of shape
        (BATCH_SIZE, NUM_INPUTS), where the first dimension index is
        inferred from the memory location (BATCH_SIZE), and the
        second dimension index is stored in the feature_indices matrix.
        The type for feature indices is int32_t.

    @param: feature_values
        A matrix of shape (BATCH_SIZE, max_active_features)
        containing the values (arity) of the corresponding
        feature index in feature_indices.
        The type for the feature value (arity) is float32.

    @param: weight_grad
        The weight gradient matrix of shape (NUM_INPUTS, output_size).
        The gradient is accumulated, i.e. it must be zero initialized
        on the first call.
        Weights must be of type float32.

    @param: bias_grad
        The bias gradient vector of shape (output_size,).
        The gradient is accumulated, i.e. it must be zero initialized
        on the first call.
        Bias values must be of type float32.

    @param: output_grad
        An output gradient matrix of shape (BATCH_SIZE, output_size).
        Output values must have type float32.
*/
void feature_transformer_slice_backward(
    const int32_t* const feature_indices,
    const float*   const feature_values,
          float*   const weight_grad,
          float*   const bias_grad,
    const float*   const output_grad
) {

    const uint32_t output_size = 256;
    const uint32_t max_active_features = 32; 
    const uint32_t output_thread_slice_size = 1;

    __shared__
          float          shared_output_grad[output_size];

    const uint32_t       block_idx                = blockIdx.x;
    const uint32_t       slice_offset             = threadIdx.x * output_thread_slice_size;

    const float*   const output_grad_slice        = output_grad + block_idx * output_size + slice_offset;
          float*   const bias_grad_slice          = bias_grad                               + slice_offset;
          float*         shared_output_grad_slice = shared_output_grad                      + slice_offset;

    const int32_t* const feature_index_row        = feature_indices + block_idx * max_active_features;
    const float*   const feature_value_row        = feature_values  + block_idx * max_active_features;

    #pragma unroll
    for (uint32_t s = 0; s < output_thread_slice_size; ++s)
    {
        shared_output_grad_slice[s] = output_grad_slice[s];
    }

    #pragma unroll
    for (uint32_t s = 0; s < output_thread_slice_size; ++s)
    {
        const float sog = shared_output_grad_slice[s];
        if (sog != 0.0f)
        {
            atomicAdd(&bias_grad_slice[s], sog);
        }
    }

    for (uint32_t k = 0; k < max_active_features; ++k)
    {
        const int32_t feature_index = feature_index_row[k];
        const float   feature_value = feature_value_row[k];
        if (feature_index != -1)
        {
            float* const weight_grad_slice = weight_grad + feature_index * output_size + slice_offset;
            #pragma unroll
            for (int s = 0; s < output_thread_slice_size; ++s)
            {
                const float sog = shared_output_grad_slice[s];
                if (sog != 0.0f)
                {
                    atomicAdd(&weight_grad_slice[s], sog * feature_value);
                }
            }
        } else break;
    }
}

/*
    num_threads = get_num_threads_for_forward(output_size)
    print('num_threads = ', num_threads)
    output_thread_slice_size = output_size // num_threads
    print('output_thread_slice_size = ', output_thread_slice_size)
    key = (max_active_features, output_size, num_threads)
    print(key)
*/

//template <uint32_t output_size, uint32_t max_active_features>
void feature_transformer_slice_forward_wrapper(
    const int32_t  batch_size,
    const int32_t* const feature_indices,
    const float*   const feature_values,
    const float*   const weight,
    const float*   const bias,
          float*   const output
) {
    const uint32_t output_size = 256;
    const uint32_t max_active_features = 32; 

    int32_t num_threads = get_num_threads_for_forward(output_size);
    int32_t output_thread_slice_size = output_size / num_threads;

    //printf("start cuda call\n");
    //feature_transformer_slice_forward<<<1, 1>>>(feature_indices, feature_values, weight, bias, output);
    feature_transformer_slice_forward<<<batch_size, num_threads>>>(feature_indices, feature_values, weight, bias, output);
}

//template <uint32_t output_size, uint32_t max_active_features>
void feature_transformer_slice_backward_wrapper(
    const int32_t  batch_size,
    const int32_t* const feature_indices,
    const float*   const feature_values,
          float*   const weight_grad,
          float*   const bias_grad,
    const float*   const output_grad
) {
    const uint32_t output_size = 256;
    const uint32_t max_active_features = 32; 

    int32_t num_threads = get_num_threads_for_backward(output_size);
    int32_t output_thread_slice_size = output_size / num_threads;

    //printf("start cuda call\n");
    feature_transformer_slice_backward<<<batch_size, num_threads>>>(feature_indices, feature_values, weight_grad, bias_grad, output_grad);
}